#include "distributed_simulator.h"

using namespace sim;
using namespace Legion;

DistributedSimulator::DSHandler
DistributedSimulator::cuda_init_task(Task const *task,
                                     std::vector<PhysicalRegion> const &regions,
                                     Context ctx, Runtime *runtime) {
  DSConfig const *config = (DSConfig *)task->args;
  DSHandler handle;
  handle.workSpaceSize = (size_t)1 * 1024 * 1024 * 1024; // 1GB work space
  handle.num_local_qubits = config->num_local_qubits;
  printf("Num_local_qubits = %lld\n", handle.num_local_qubits);
  custatevecCreate(&handle.statevec);
  {
    // allocate memory for workspace
    Memory gpu_mem = Machine::MemoryQuery(Machine::get_machine())
                         .only_kind(Memory::GPU_FB_MEM)
                         .best_affinity_to(task->target_proc)
                         .first();
    Realm::Rect<1, coord_t> bounds(
        Realm::Point<1, coord_t>(0),
        Realm::Point<1, coord_t>(handle.workSpaceSize - 1));
    std::vector<size_t> field_sizes;
    field_sizes.push_back(sizeof(char));
    Realm::RegionInstance workspaceInst;
    Realm::RegionInstance::create_instance(workspaceInst, gpu_mem, bounds,
                                           field_sizes, 0,
                                           Realm::ProfilingRequestSet())
        .wait();
    handle.workSpace = workspaceInst.pointer_untyped(0, sizeof(char));
  }
  handle.ncclComm = nullptr;
  handle.vecDataType = config->state_vec_data_type;
  return handle;
}

void DistributedSimulator::sv_init_task(
    Task const *task, std::vector<PhysicalRegion> const &regions, Context ctx,
    Runtime *runtime) {
  // TODO: implement this function
  printf("SV Init...\n");
  return;
}

void DistributedSimulator::sv_comp_task(
    Task const *task, std::vector<PhysicalRegion> const &regions, Context ctx,
    Runtime *runtime) {
  GateInfo const *info = (GateInfo *)task->args;
  FusedGate* fgates = (FusedGate*) info->fgates;
  printf("num_target: %d, %d, %d, %d\n", fgates[0].num_target, fgates[1].num_target, fgates[2].num_target, fgates[3].num_target);
  // if (info == nullptr) return;
  DSHandler const *handler = (DSHandler *)task->local_args;
  
  assert(handler->vecDataType == DT_FLOAT_COMPLEX || handler->vecDataType == DT_DOUBLE_COMPLEX);
  hipDataType data_type = handler->vecDataType == DT_FLOAT ? HIP_C_32F : HIP_C_64F;
  custatevecComputeType_t compute_type = handler->vecDataType == DT_FLOAT ? CUSTATEVEC_COMPUTE_32F : CUSTATEVEC_COMPUTE_64F;
  GenericTensorAccessorW state_vector = helperGetGenericTensorAccessorWO(
      handler->vecDataType, regions[0], task->regions[0], FID_DATA, ctx, runtime);

  // KernelGate* gates = (KernelGate*) info->kgates;

  // printf("%d Batched Tasks: fusion task 1: %d targets, targetqubit %d\n", info->num_tasks, gates[0].num_target, gates[0].target[0]);
  // printf("%d Batched Tasks: targetQubit %d, %d\n", info->num_tasks, gates[0].targetQubit, gates[20].targetQubit);
  

  // if(info->gtype == SHM) {
  //   KernelGate* kgate = (KernelGate*) info->kgates;
  //   printf("kernel target %d\n", kgate[0].targetQubit);
  // }
  // else if(info->gtype == FUSED) {
  //   FusedGate* gates = (FusedGate*) info->fgates;
  //   printf("Fusion Kernel %p\n", gates);
  //   printf("%d, Fusion Kernel Target %d, %d\n", info->num_batched_gates, gates[0].num_target, gates[0].target[0]);
  //   return;
  // }
  
  // FUSED Gates
  unsigned const nIndexBits = handler->num_local_qubits;

  for (int i = 1; i < 4; i++) {
    unsigned const nTargets = fgates[i].num_target;
    unsigned const nControls = fgates[i].num_control;
    int const adjoint = 0;
    std::vector<int> targets;
    std::vector<int> controls;

  
    for (int k = 0; k < nTargets; k++) {
      targets.push_back(k);
    }

    //   // apply gate
    custatevecApplyMatrix(
        /* custatevecHandle_t */ handler->statevec,
        /* void* */ state_vector.get_void_ptr(),
        /* hipDataType */ data_type,
        /* const uint32_t */ nIndexBits,
        /* const void* */ fgates[i].matrix,
        /* hipDataType */ data_type,
        /* custatevecMatrixLayout_t */ CUSTATEVEC_MATRIX_LAYOUT_ROW,
        /* const int32_t */ adjoint,
        /* const int32_t* */ targets.data(),
        /* const uint32_t */ nTargets,
        /* const int32_t* */ controls.data(),
        /* const int32_t* */ nullptr,
        /* const uint32_t */ nControls,
        /* custatevecComputeType_t */ compute_type,
        /* void* */ handler->workSpace,
        /* size_t */ handler->workSpaceSize);
  }
  
  // for (int gate_idx=0; gate_idx < info->num_batched_gates; gate_idx++){
  //   if (info->gtype == SHM) break;
  //   // TODO: get target & control qubit idx from current perm[]
  //   Gate<qreal> gate = info->gates[gate_idx];
  //   std::vector<int> targets;
  //   std::vector<int> controls;
    
  //   unsigned const nTargets = gate.num_target;
  //   unsigned const nControls = gate.num_control;
  //   int const adjoint = 0;
  //   // TODO: check if targets should be ordered
  //   printf("Targets: [");
  //   for (int i = 0; i < gate.num_target; i++) {
  //     int idx = 0;
  //     while (info->permutation[idx] != gate.target[i])
  //       idx++;
  //     targets.push_back(idx);
  //     printf("(%d, %d) ", gate.target[i], idx);
  //   }
  //   printf("]\n");

  //   for (int i = 0; i < gate.num_control; i++) {
  //     int idx = 0;
  //     while (info->permutation[idx] != gate.target[i])
  //       idx++;
  //     controls.push_back(idx);
  //   }

  //   // apply gate
  //   custatevecApplyMatrix(
  //       /* custatevecHandle_t */ handler->statevec,
  //       /* void* */ state_vector.get_void_ptr(),
  //       /* hipDataType */ data_type,
  //       /* const uint32_t */ nIndexBits,
  //       /* const void* */ gate.matrix.data(),
  //       /* hipDataType */ data_type,
  //       /* custatevecMatrixLayout_t */ CUSTATEVEC_MATRIX_LAYOUT_ROW,
  //       /* const int32_t */ adjoint,
  //       /* const int32_t* */ targets.data(),
  //       /* const uint32_t */ nTargets,
  //       /* const int32_t* */ controls.data(),
  //       /* const int32_t* */ nullptr,
  //       /* const uint32_t */ nControls,
  //       /* custatevecComputeType_t */ compute_type,
  //       /* void* */ handler->workSpace,
  //       /* size_t */ handler->workSpaceSize);

  //   }
  
}
